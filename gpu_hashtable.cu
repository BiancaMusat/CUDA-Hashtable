#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include <stdint.h>

#include "gpu_hashtable.hpp"

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
	hash_size = size; // actual size of hashtable
	num_entries = 0;  // number of occupied slots
	hipMalloc((void **) &hashtable, size * sizeof(entry));
	hipMemset(hashtable, KEY_INVALID, size * sizeof(entry));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	hipFree(hashtable);
}

/* Hash function used by hashtable
 */
__device__ uint32_t hash_func(int data, int limit) {
	return ((long)abs(data) * 105359939) % 1685759167 % limit;
}

/* resize function that will be run by GPU
 */
__global__ void resize(GpuHashTable::entry *hashtable, GpuHashTable::entry *new_hash,
						int hash_size, int numBucketsReshape) {
	/* each thread will copy one element from hashtable to new_hash */
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < hash_size) {
		if (hashtable[tid].key == KEY_INVALID)
			return;
		/* rehash each key */
		uint32_t key = hash_func(hashtable[tid].key, numBucketsReshape);
		while (true) {
			/* find empty slot and add pair */
			uint32_t prev = atomicCAS(&new_hash[key].key, KEY_INVALID, hashtable[tid].key);
			if (prev == hashtable[tid].key || prev == KEY_INVALID) {
				new_hash[key].value = hashtable[tid].value;
				break;
			}
			key++;
			key %= numBucketsReshape;
		}
	}
}
/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	uint32_t block_size = 100;
	uint32_t blocks_no = hash_size / block_size;
	if (hash_size % block_size)
		++blocks_no;
	struct entry *new_hash;
	/* alloc new hash */
	hipMalloc((void **) &new_hash, numBucketsReshape * sizeof(entry));
	hipMemset(new_hash, KEY_INVALID, numBucketsReshape * sizeof(entry));
	resize<<<blocks_no, block_size>>>(hashtable, new_hash, hash_size, numBucketsReshape);
	hipDeviceSynchronize();
	hipFree(hashtable);
	hashtable = new_hash;
	hash_size = numBucketsReshape;
}

/* insert function that will be run by GPU
 */
__global__ void insert(GpuHashTable::entry *hashtable, int hash_size,
						int *keys, int* values, int numKeys) {
	/* each thread will insert one element into hashtable */
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < numKeys) {
		/* compute hash for key */
		uint32_t key = hash_func(keys[tid], hash_size);
		while (true) {
			/* find empty spot or update value if the key already exists */
			uint32_t prev = atomicCAS(&hashtable[key].key, KEY_INVALID, keys[tid]);
			if (prev == keys[tid] || prev == KEY_INVALID) {
				hashtable[key].value = values[tid];
				return;
			}
			key++;
			key %= hash_size;
		}
	}
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int *new_values;
	/* compute number of entries before calling insert in order to perform
	 * reshape if needed
	 */
	new_values = getBatch(keys, numKeys);
	for (int i = 0; i < numKeys; i++)
		if (new_values[i] == KEY_INVALID)
			num_entries++;
	if ((float)(num_entries) / hash_size >= 0.9)
		reshape(num_entries + (int)(0.1 * num_entries));

	uint32_t block_size = 100;
	uint32_t blocks_no = numKeys / block_size;
	if (numKeys % block_size)
		++blocks_no;
	int *dev_keys = 0;
	int *dev_values = 0;
	/* alloc memory for GPU and copy keys and values arrays into GPU memory */
	hipMalloc((void **) &dev_keys, numKeys * sizeof(int));
	hipMalloc((void **) &dev_values, numKeys * sizeof(int));
	hipMemcpy(dev_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	insert<<<blocks_no, block_size>>>(hashtable, hash_size, dev_keys, dev_values, numKeys);
	hipDeviceSynchronize();
	hipFree(dev_keys);
	hipFree(dev_values);
	free(new_values);
	return true;
}

/* get function that will be run by GPU
 */
__global__ void get(GpuHashTable::entry *hashtable, int hash_size,
						int *keys, int *values, int numKeys) {
	/* each thread will add to the result array one element from hashtable
	 * corresponding to one key form keys array
	 */
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < numKeys) {
		/* compute hash for key */
		uint32_t key = hash_func(keys[tid], hash_size);
		while (true) {
			if (hashtable[key].key == keys[tid]) {
				values[tid] = hashtable[key].value;
				break;
			}
			if (hashtable[key].key == KEY_INVALID) {
				values[tid] = KEY_INVALID;
				break;
			}
			key++;
			key %= hash_size;
		}
	}
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *results = (int *)malloc(numKeys * sizeof(int));
	uint32_t block_size = 100;
	uint32_t blocks_no = numKeys / block_size;
	if (numKeys % block_size)
		++blocks_no;
	int *dev_keys = 0;
	int *dev_values = 0;
	/* alloc memory for GPU and copy keys and values arrays into GPU memory */
	hipMalloc((void **) &dev_keys, numKeys * sizeof(int));
	hipMalloc((void **) &dev_values, numKeys * sizeof(int));
	hipMemcpy(dev_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(dev_values, KEY_INVALID, numKeys * sizeof(int));
	get<<<blocks_no, block_size>>>(hashtable, hash_size, dev_keys, dev_values, numKeys);
	hipDeviceSynchronize();
	/* copy vallues array from GPU memory into results array (CPU memory) */
	hipMemcpy(results, dev_values, numKeys * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_keys);
	hipFree(dev_values);
	return results;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	return (float)num_entries / hash_size; // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#define HASH_DESTROY GpuHashTable.~GpuHashTable();

#include "test_map.cpp"
